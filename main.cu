#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include<GL/glew.h>

#include "Globals.h"
#include <time.h>
#include <thread>
#include "Helpers.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int isPaused = 0;

unsigned int world_width;
unsigned int world_height;

int *cells;
int *d_ocells;
int *d_icells;
struct uchar4 *d_dst;

#define BUFFER_DATA(i) ((char *)0 + i)

//OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex;
//Source image on the host side
uchar4 *h_Src;
//Size of displayed image
int imageW, imageH;


void RunGameOfLifeKernel(int *i_cells, int *o_cells, int width, int height, uchar4* dst);


////////////////////////////////////////////////////////////////////////////////
//OPENGL FUNCTIONS
////////////////////////////////////////////////////////////////////////////////
void displayFunc(void)
{
	
	d_dst = NULL;

	cudaGLMapBufferObject((void**)&d_dst, gl_PBO);

	RunGameOfLifeKernel(d_icells, d_ocells, world_width, world_height, d_dst);

	cudaGLUnmapBufferObject(gl_PBO);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));
	glBegin(GL_TRIANGLES);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(-1.0f, -1.0f);
	glTexCoord2f(2.0f, 0.0f);
	glVertex2f(3.0f, -1.0f);
	glTexCoord2f(0.0f, 2.0f);
	glVertex2f(-1.0f, 3.0f);
	glEnd();
	glutSwapBuffers();
} // displayFunc

void idleFunc()
{
	if (!isPaused)
	{
		glutPostRedisplay();
		std::chrono::milliseconds dur(1000 / FPS);
		std::this_thread::sleep_for(dur);
	}
}
void closeFunc()
{
	gpuErrchk(hipFree(d_ocells));
	gpuErrchk(hipFree(d_icells));
	free(h_Src);
	free(cells);
	printf("Closed\n");
}
void keyboardFunc(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'p':
		isPaused = isPaused == 0 ? 1 : 0;
		break;
	default:
		break;
	}
}
bool initGL(int *argc, char **argv)
{
	imageH = world_height;
	imageW = world_width;

	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(imageW, imageH);
	glutCreateWindow(argv[0]);
	printf("Loading extensions: %s\n", (char*)glewGetErrorString(glewInit()));
	if (!glewIsSupported(
		"GL_VERSION_2_0 "
		"GL_ARB_pixel_buffer_object "
		"GL_EXT_framebuffer_object "
		)){
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}
	printf("OpenGL window created.\n");

	printf("Creating GL texture...\n");
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &gl_Tex);
	glBindTexture(GL_TEXTURE_2D, gl_Tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, imageW, imageH, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);
	printf("Texture created.\n");

	printf("Creating PBO...\n");
	glGenBuffers(1, &gl_PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, imageW * imageH * 4, h_Src, GL_STREAM_COPY);
	//While a PBO is registered to CUDA, it can't be used 
	//as the destination for OpenGL drawing calls.
	//But in our particular case OpenGL is only used 
	//to display the content of the PBO, specified by CUDA kernels,
	//so we need to register/unregister it only once.
	cudaGLRegisterBufferObject(gl_PBO);
	printf("PBO created.\n");

	glutDisplayFunc(displayFunc);
	glutIdleFunc(idleFunc);
	glutCloseFunc(closeFunc);
	glutKeyboardFunc(keyboardFunc);
	return true;
}//initgl

//end opengl functions


///////////////////////////////////////////////////////////////////////////////
//GPU FUNCTIONS
///////////////////////////////////////////////////////////////////////////////

__device__ int CountAliveCells(int *i_cells, int idx, int width, int height)
{
	int alive = 0;

	int posY = floorf(idx / width);
	int posX = idx % width;

	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			int currPosX = (posX + i) % width;
			int currPosY = (posY + j) % height;

			if (currPosX < 0)
			{
				currPosX = width + currPosX;
			}
			if (currPosY < 0)
			{
				currPosY = height + currPosY;
			}

			int neigh = currPosY * width + currPosX;

			if (i == 0 && j == 0)
				continue;
			if (i_cells[neigh] == 1)
				alive++;
		}
	}
	return alive;
}
__global__ void UpdateGrid(int *i_cells, int *o_cells, struct uchar4 *dst)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	i_cells[idx] = o_cells[idx];

	//assign color
	dst[idx].x = i_cells[idx] * 255;
	dst[idx].y = i_cells[idx] * 255;
	dst[idx].z = i_cells[idx] * 255;
}

__global__ void CalcNextGeneration(int *i_cells, int *o_cells, int width, int height, uchar4 *dst)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= width*height)
		return;

	int neighCount = CountAliveCells(i_cells, idx, width, height);


	if (neighCount == 3 ||
		(neighCount == 2 && i_cells[idx] == 1))
		o_cells[idx] = 1;
	else
		o_cells[idx] = 0;

	__syncthreads();

}
//end gpu functions

void RunGameOfLifeKernel(int *i_cells, int *o_cells, int width, int height, uchar4* dst)
{
	int size = width*height;
	dim3 threads(THREADS_PER_BLOCK, 1, 1);
	dim3 blocks(ceil(((float)size) / (float)(THREADS_PER_BLOCK)), 1, 1);
	CalcNextGeneration<<<blocks, threads >>>(i_cells, o_cells, width, height, dst);
	gpuErrchk(hipGetLastError());
	UpdateGrid << <blocks, threads >> >(i_cells, o_cells, dst);
	gpuErrchk(hipGetLastError());
}

void initWorld(int gridWidth, int gridHeight, char* filename)
{
	int *grid = (int*)malloc(gridWidth*gridHeight * sizeof(int));

	if (LoadGridFromFile(gridWidth, gridHeight, grid, filename) == 1)
	{
		printf("Error occured!\n Not proper data in file: %s", filename);
		exit(EXIT_FAILURE);
	}

	cells = (int*)calloc(world_width*world_height, sizeof(int));
	//printCells(grid, gridWidth, gridHeight);
	int offsetX = world_height / 2 - gridHeight / 2;
	int offsetY = world_width / 2 - gridWidth / 2;

	for (int i = 0; i < gridHeight; i++)
	{
		for (int j = 0; j < gridWidth; j++)
		{
			int posX = (offsetX + i) % world_width;
			int posY = (offsetY + j) % world_height;
			cells[posX*world_width + posY] = grid[i*gridWidth + j];
		}
	}
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{	
	int size;
	world_width = 500;
	world_height = 500;
	int gridWidth;
	int gridHeight;
	switch (argc)
	{ 
	case 4: //random map with provided grid dimensions
		if (strcmp(argv[1], "-w")==0)
		{
			world_width = atoi(argv[2]);
			world_height = atoi(argv[3]);
			if (world_width < 5 || world_height < 5)
			{
				usage(argv[0]);
			}
		}
		else
		{
			usage(argv[0]);
		}
	case 1: //init random map whether with initial size or with default (1000x1000)
		size = world_height * world_width;
		cells = (int*)malloc(size* sizeof(int));
		randomMap(cells, world_width, world_height);
		break;
	case 8:
		world_width = atoi(argv[2]);
		world_height = atoi(argv[3]);
		gridWidth = atoi(argv[5]);
		gridHeight = atoi(argv[6]);
		size = world_height * world_width;

		initWorld(gridWidth, gridHeight, argv[7]);
		break;
	default:
		usage(argv[0]);
		break;
	}
	printf("Cells grid created...\n");
	gpuErrchk(hipMalloc(&d_icells, size * sizeof(int)));
	gpuErrchk(hipMalloc(&d_ocells, size*sizeof(int)));
	
	gpuErrchk(hipMemcpy(d_icells, cells, size*sizeof(int), hipMemcpyHostToDevice));

	initGL(&argc, argv);
	printf("\nStarted\n");	
	printf("Controls:\n");
	printf("p - to pause/resume (after pausing to see next step click LPM)\n");

	glutMainLoop();
	printf("Ended\n");
}