#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
	Max dimensions of the world: 6000 x 5500
	Parameters: Intel Core i5-2500K 3.30GHz
				8GB RAM
				NVIDIA GeForce GTX 560 Ti 
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include<GL/glew.h>

#include "Globals.h"
#include <time.h>
#include <thread>
#include "Helpers.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int isPaused = 0;

unsigned int world_width;
unsigned int world_height;

int *cells;
int *d_ocells;
int *d_icells;
struct uchar4 *d_dst;

#define BUFFER_DATA(i) ((char *)0 + i)

//OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex;
//Source image on the host side
uchar4 *h_Src;
//Size of displayed image
int imageW, imageH;

int offsetX = 0, offsetY = 0; //where camera starts rendering
float scale = 1;
float alreadyScaled = 1.2;
float slow = 1;
void RunGameOfLifeKernel(int *i_cells, int *o_cells, int width, int height, uchar4* dst);
float milliseconds =1.0;

////////////////////////////////////////////////////////////////////////////////
//OPENGL FUNCTIONS
////////////////////////////////////////////////////////////////////////////////
void displayFunc(void)
{
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	d_dst = NULL;

	cudaGLMapBufferObject((void**)&d_dst, gl_PBO);

	hipEventRecord(start);
	RunGameOfLifeKernel(d_icells, d_ocells, world_width, world_height, d_dst);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	cudaGLUnmapBufferObject(gl_PBO);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));
	glBegin(GL_TRIANGLES);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(-1.0f, -1.0f);
	glTexCoord2f(2.0f, 0.0f);
	glVertex2f(3.0f, -1.0f);
	glTexCoord2f(0.0f, 2.0f);
	glVertex2f(-1.0f, 3.0f);
	glEnd();

	glScalef(scale, scale, 1);
	scale = 1;
	glutSwapBuffers();
} // displayFunc

void idleFunc()
{
	glutPostRedisplay();
	int toSleep = (1000 - (int)milliseconds) / FPS*slow;
	int fps = 1000 / toSleep;
	char buff[50];
	sprintf(buff, "FPS: %i; X: %i; Y: %i; Slow: %.2f", fps, offsetX, offsetY, slow);
	glutSetWindowTitle( buff);
	std::chrono::milliseconds dur(toSleep);
	std::this_thread::sleep_for(dur);
}
void closeFunc()
{
	gpuErrchk(hipFree(d_ocells));
	gpuErrchk(hipFree(d_icells));
	free(h_Src);
	free(cells);
	printf("Closed\n");
}
void keyboardFunc(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'p':
		isPaused = isPaused == 0 ? 1 : 0;
		break;
	case 'w':
			offsetY = (offsetY += STEP_SIZE)%world_width;
		break;
	case 's':
		offsetY = (offsetY -= STEP_SIZE) % world_width;
		break;
	case 'a':
			offsetX = (offsetX -= STEP_SIZE)%world_height;
		break;
	case 'd':	
		offsetX = (offsetX += STEP_SIZE) % world_height;
		break;
	case 'm':
		slow += 0.1;
		break;
	case 'n':
		if (slow > 0.2)
		{
			slow -= 0.1;
		}
		
		break;
	//case 'z':
	//	scale = 1.1;
	//	alreadyScaled *= 1.1;
	//	break;
	//case 'x':
	//	alreadyScaled *= 0.9;
	//	scale = 0.9;
			
	default:
		break;
	}
}
bool initGL(int *argc, char **argv)
{
	imageH = MAX_WINDOW_HEIGHT;
	imageW = MAX_WINDOW_WIDTH;
	if (world_height <= MAX_WINDOW_HEIGHT)
		imageH = world_height;
	if (world_width <= MAX_WINDOW_WIDTH)
		imageW = world_width;

	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(imageW, imageH);
	glutCreateWindow(argv[0]);
	printf("Loading extensions: %s\n", (char*)glewGetErrorString(glewInit()));
	if (!glewIsSupported(
		"GL_VERSION_2_0 "
		"GL_ARB_pixel_buffer_object "
		"GL_EXT_framebuffer_object "
		)){
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}
	printf("OpenGL window created.\n");

	printf("Creating GL texture...\n");
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &gl_Tex);
	glBindTexture(GL_TEXTURE_2D, gl_Tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, imageW, imageH, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);
	printf("Texture created.\n");

	printf("Creating PBO...\n");
	glGenBuffers(1, &gl_PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, imageW * imageH * 4, h_Src, GL_STREAM_COPY);
	//While a PBO is registered to CUDA, it can't be used 
	//as the destination for OpenGL drawing calls.
	//But in our particular case OpenGL is only used 
	//to display the content of the PBO, specified by CUDA kernels,
	//so we need to register/unregister it only once.
	cudaGLRegisterBufferObject(gl_PBO);
	printf("PBO created.\n");

	glutDisplayFunc(displayFunc);
	glutIdleFunc(idleFunc);
	glutCloseFunc(closeFunc);
	glutKeyboardFunc(keyboardFunc);
	return true;
}//initgl

//end opengl functions


///////////////////////////////////////////////////////////////////////////////
//GPU FUNCTIONS
///////////////////////////////////////////////////////////////////////////////

__device__ int CountAliveCells(int *i_cells, int idx, int width, int height)
{
	int alive = 0;

	int posY = floorf(idx / width);
	int posX = idx % width;

	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			int currPosX = (posX + i) % width;
			int currPosY = (posY + j) % height;

			if (currPosX < 0)
			{
				currPosX = width + currPosX;
			}
			if (currPosY < 0)
			{
				currPosY = height + currPosY;
			}

			int neigh = currPosY * width + currPosX;

			if (i == 0 && j == 0)
				continue;
			if (i_cells[neigh] == 1)
				alive++;
		}
	}
	return alive;
}
__global__ void UpdateGrid(int offX, int offY ,int *i_cells, struct uchar4 *dst, int width, int height, int worldW, int worldH)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x; //position in visible grid
	int posX =offX + idx%width;
	int posY =offY + idx / width;


	int cellIdx = posX*worldW + posY; //calc position of cell in grid
	if ( idx < width*height)
	{
		//assign color
		dst[idx].x = i_cells[cellIdx] * 255;
		dst[idx].y = i_cells[cellIdx] * 255;
		dst[idx].z = i_cells[cellIdx] * 255;
	}
}

__global__ void CalcNextGeneration(int *i_cells, int *o_cells, int width, int height)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= width*height)
		return;

	int neighCount = CountAliveCells(i_cells, idx, width, height);


	if (neighCount == 3 ||
		(neighCount == 2 && i_cells[idx] == 1))
		o_cells[idx] = 1;
	else
		o_cells[idx] = 0;

	__syncthreads();

}
__global__ void SyncCells(int *i_cells, int *o_cells, int size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= size)
		return;
	i_cells[idx] = o_cells[idx];
}
//end gpu functions

void RunGameOfLifeKernel(int *i_cells, int *o_cells, int worldW, int worldH, uchar4* dst)
{
	int worldSize = worldW*worldH;
	dim3 threads(THREADS_PER_BLOCK, 1, 1);
	dim3 blocks(ceil(((float)worldSize) / (float)(THREADS_PER_BLOCK)), 1, 1);
	if (!isPaused)
	{
		CalcNextGeneration << <blocks, threads >> >(i_cells, o_cells, worldW, worldH);
		gpuErrchk(hipGetLastError());

		SyncCells << <blocks, threads >> >(i_cells, o_cells, worldSize); //sync cells betweeen threads
		gpuErrchk(hipGetLastError());
	}
	dim3 threadsGrid(THREADS_PER_BLOCK, 1, 1);
	dim3 blocksGrid(ceil(((float)imageW*imageH) / (float)(THREADS_PER_BLOCK)), 1, 1);
	UpdateGrid << <blocksGrid, threadsGrid >> >(offsetX, offsetY, i_cells, dst, imageW, imageH, worldW, worldH); //draw stuff
	gpuErrchk(hipGetLastError());
}

void initWorld(int gridWidth, int gridHeight, char* filename)
{
	int *grid = (int*)malloc(gridWidth*gridHeight * sizeof(int));

	if (LoadGridFromFile(gridWidth, gridHeight, grid, filename) == 1)
	{
		printf("Error occured!\n Not proper data in file: %s", filename);
		exit(EXIT_FAILURE);
	}

	cells = (int*)calloc(world_width*world_height, sizeof(int));
	//printCells(grid, gridWidth, gridHeight);
	int offsetX = 0;// world_height / 2 - gridHeight / 2;
	int offsetY = 0;// world_width / 2 - gridWidth / 2;

	for (int i = 0; i < gridHeight; i++)
	{
		for (int j = 0; j < gridWidth; j++)
		{
			int posX = (offsetX + i) % world_width;
			int posY = (offsetY + j) % world_height;
			cells[posX*world_width + posY] = grid[i*gridWidth + j];
		}
	}
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{	
	int size;
	world_width = 500;
	world_height = 500;
	int gridWidth;
	int gridHeight;
	switch (argc)
	{ 
	case 4: //random map with provided grid dimensions
		if (strcmp(argv[1], "-w")==0)
		{
			world_width = atoi(argv[2]);
			world_height = atoi(argv[3]);
			if (world_width < 5 || world_height < 5)
			{
				usage(argv[0]);
			}
		}
		else
		{
			usage(argv[0]);
		}
	case 1: //init random map whether with initial size or with default (1000x1000)
		size = world_height * world_width;
		cells = (int*)malloc(size* sizeof(int));
		randomMap(cells, world_width, world_height);
		break;
	case 8:
		world_width = atoi(argv[2]);
		world_height = atoi(argv[3]);
		gridWidth = atoi(argv[5]);
		gridHeight = atoi(argv[6]);
		size = world_height * world_width;

		initWorld(gridWidth, gridHeight, argv[7]);
		break;
	default:
		usage(argv[0]);
		break;
	}
	printf("Cells grid created...\n");
	gpuErrchk(hipMalloc(&d_icells, size * sizeof(int)));
	gpuErrchk(hipMalloc(&d_ocells, size * sizeof(int)));
	
	gpuErrchk(hipMemcpy(d_icells, cells, size*sizeof(int), hipMemcpyHostToDevice));

	initGL(&argc, argv);
	printf("\nStarted\n");	
	printf("Controls:\n");
	printf("p - to pause/resume\n");
	printf("w/s/a/d - to move on the map\n");
	printf("m - slow animation\n");
	printf("n - speed up animation\n");
	//printf("z - zoom out\n");
	//printf("x - zoom in\n");

	glutMainLoop();
	printf("Ended\n");
}