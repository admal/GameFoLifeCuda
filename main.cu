#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include<GL/glew.h>

#include "Globals.h"
#include <time.h>
#include <thread>
#include "LoadingFiles.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


unsigned int world_width;
unsigned int world_height;

int *cells;
int *d_ocells;
int *d_icells;
struct uchar4 *d_dst;

//TMP
void printCells(int* cells, int width, int height)
{
	int size = width*height;
	for (int i = 0; i < size; i++)
	{
		if (i%width == 0)
			printf("\n");
		printf("%i ", cells[i]);
	}
	printf("\n");
}
//end tmp
void randomMap(int *i_cells, int width, int height)
{
	srand(time(NULL));
	for (int i = 0; i < width*height; i++)
	{
		i_cells[i] = rand()%100 >= (100 -SCREEN_COVERAGE) ? 1 : 0;
	}
}

void RunGameOfLifeKernel(int *i_cells, int *o_cells, int width, int height, uchar4* dst);

////////////////////////////////////////////////////////////////////////////////
//OPENGL FUNCTIONS
////////////////////////////////////////////////////////////////////////////////

#define BUFFER_DATA(i) ((char *)0 + i)

//OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex;
//Source image on the host side
uchar4 *h_Src;

//Original image width and height
int imageW, imageH;

// Timer ID
//unsigned int hTimer;

void displayFunc(void)
{
	d_dst = NULL;

	cudaGLMapBufferObject((void**)&d_dst, gl_PBO);

	RunGameOfLifeKernel(d_icells, d_ocells, world_width, world_height, d_dst);

	cudaGLUnmapBufferObject(gl_PBO);

	//glPushMatrix();  //begin scaling
	//glScalef(10, 10, 10);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));
	glBegin(GL_TRIANGLES);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(-1.0f, -1.0f);
	glTexCoord2f(2.0f, 0.0f);
	glVertex2f(3.0f, -1.0f);
	glTexCoord2f(0.0f, 2.0f);
	glVertex2f(-1.0f, 3.0f);
	glEnd();
	//glPopMatrix();
	glutSwapBuffers();
} // displayFunc

void idleFunc()
{
	glutPostRedisplay();
	std::chrono::milliseconds dur(1000 / FPS);
	std::this_thread::sleep_for(dur);
}

bool initGL(int *argc, char **argv)
{
	imageH = world_height;
	imageW = world_width;


	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(imageW, imageH);
	glutCreateWindow(argv[0]);
	printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));
	if (!glewIsSupported(
		"GL_VERSION_2_0 "
		"GL_ARB_pixel_buffer_object "
		"GL_EXT_framebuffer_object "
		)){
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}
	printf("OpenGL window created.\n");

	printf("Creating GL texture...\n");
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &gl_Tex);
	glBindTexture(GL_TEXTURE_2D, gl_Tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, imageW, imageH, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);
	printf("Texture created.\n");

	printf("Creating PBO...\n");
	glGenBuffers(1, &gl_PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, imageW * imageH * 4, h_Src, GL_STREAM_COPY);
	//While a PBO is registered to CUDA, it can't be used 
	//as the destination for OpenGL drawing calls.
	//But in our particular case OpenGL is only used 
	//to display the content of the PBO, specified by CUDA kernels,
	//so we need to register/unregister it only once.
	cudaGLRegisterBufferObject(gl_PBO);
	printf("PBO created.\n");

	glutDisplayFunc(displayFunc);
	glutIdleFunc(idleFunc);
	return true;
}

//end opengl functions




///////////////////////////////////////////////////////////////////////////////
//GPU FUNCTIONS
///////////////////////////////////////////////////////////////////////////////
__device__ int CountAliveCells(int *i_cells, int idx, int width, int height)
{
	int alive = 0;

	int posY = floorf(idx/width);
	//int posX = idx - posY*width;
	int posX = idx % width;

	//if (idx == 4)
	//	printf("Idx: 4; x = %i, y = %i;\n", posX, posY);

	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			int currPosX = (posX + i) % width;
			int currPosY = (posY + j) % height;

			if (currPosX < 0)
			{
				currPosX = width + currPosX - 1;
			}
			if (currPosY < 0)
			{
				currPosY = height + currPosY - 1;
			}

			int neigh = currPosY * width + currPosX;
			
			//if (idx == 4)
			//	printf("neighIdx: %i\n", neigh);

			if (i == 0 && j == 0)
				continue;
			if (i_cells[neigh] == 1)
				alive++;
		}
	}
	return alive;
}

__global__ void CalcNextGeneration(int *i_cells, int *o_cells, int width, int height, uchar4 *dst)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= width*height)
		return;
	if (idx == 24)
	{
		printf("24 cell started...\n");
		printf("init: %i\n", i_cells[idx]);
		printf("Idx: %i;Neigh: %i; \n", idx, CountAliveCells(i_cells, idx, width, height));
	}

	if (CountAliveCells(i_cells, idx,width,height) == 3|| 
		(CountAliveCells(i_cells, idx,width,height) == 2 && i_cells[idx]==1))
		o_cells[idx] = 1;
	else
		o_cells[idx] = 0;
	
	__syncthreads();
	i_cells[idx] = o_cells[idx];
	
	//assign color
	dst[idx].x = i_cells[idx] * 255;
	dst[idx].y = i_cells[idx] * 255;
	dst[idx].z = i_cells[idx] * 255;
	//if (idx == 24)
	//	printf("color: %i\n", dst[idx].x);
}

//end gpu functions

void RunGameOfLifeKernel(int *i_cells, int *o_cells, int width, int height, uchar4* dst)
{
	int size = width*height;
	dim3 threads(THREADS_PER_BLOCK, 1, 1);
	dim3 blocks(ceil(size / THREADS_PER_BLOCK), 1, 1);

	CalcNextGeneration<<<blocks, threads >>>(i_cells, o_cells, width, height, dst);
	gpuErrchk(hipGetLastError());
}


void usage(char* name)
{
	fprintf(stderr, "%s [-w worldW worldH [-f width height filename]]\n", name);
	fprintf(stderr, "worldW - world width, worldH - world height \n");
	fprintf(stderr, "width, height - dimensions of initial the grid (not of world!)\n");
	fprintf(stderr, "filename - name of provided initial part of world\n");
	exit(EXIT_FAILURE);
}

void initWorld(int gridWidth, int gridHeight, char* filename)
{
	int *grid = (int*)malloc(gridWidth*gridHeight * sizeof(int));

	if ( LoadGridFromFile(gridWidth, gridHeight, grid, filename) == 1)
	{
		printf("Error occured!\n Not proper data in file: %s", filename);
		exit(EXIT_FAILURE);
	}


	cells = (int*)calloc(world_width*world_height, sizeof(int));
	printCells(grid, gridWidth,gridHeight);
	int offsetX = 0;// world_height / 2 - gridHeight / 2;
	int offsetY = 0;// world_width / 2 - gridWidth / 2;

	for (int i = 0; i < gridWidth; i++)
	{
		for (int j = 0; j < gridHeight; j++)
		{
			int posX = offsetX + i;
			int posY = offsetY + j;
			cells[posX*world_width + posY] = grid[i*gridWidth + j];
		}
	}

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{	
	int size;
	world_width = 1000;
	world_height = 1000;
	int gridWidth;
	int gridHeight;
	printf("argc = %i\n", argc);
	switch (argc)
	{ 
	case 4: //random map with provided grid
		printf("argv[1] = %s\n", argv[1]);
		if (strcmp(argv[1], "-w")==0)
		{
			world_width = atoi(argv[2]);
			world_height = atoi(argv[3]);

			if (world_width < 5 || world_height < 5)
			{
				printf("Error line 356\n");
				usage(argv[0]);
			}
		}
		else
		{
			printf("Error line 362\n");
			usage(argv[0]);
		}
	case 1: //init random map whether with initial size or with default
		size = world_height * world_width;
		cells = (int*)malloc(size* sizeof(int));
		randomMap(cells, world_width, world_height);
		break;
	case 8:
		world_width = atoi(argv[2]);
		world_height = atoi(argv[3]);
		gridWidth = atoi(argv[5]);
		gridHeight = atoi(argv[6]);
		size = world_height * world_width;
		initWorld(gridWidth, gridHeight, argv[7]);
		break;
	default:
		usage(argv[0]);
		break;
	}

	gpuErrchk(hipMalloc(&d_icells, size * sizeof(int)));
	gpuErrchk(hipMalloc(&d_ocells, size*sizeof(int)));
	gpuErrchk(hipMemcpy(d_icells, cells, size*sizeof(int), hipMemcpyHostToDevice));
	initGL(&argc, argv);
	printf("\nStarted\n");	

	glutMainLoop();
}